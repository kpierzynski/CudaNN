﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Mnist.h"
#include "Network.h"
#include "Linear.h"
#include "MSE.h"
#include "Tanh.h"
#include "Plants.h"

#define BATCH_SIZE 32

int main()
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?\r\n");
		return 1;
	}

	Plants plants("C:\\PLANTS\\color", BATCH_SIZE, 360);
	Network net2;
	net2.addLayer(new Linear(256 * 256 * 3, 512, BATCH_SIZE));
	net2.addLayer(new Tanh(512, BATCH_SIZE));

	net2.addLayer(new Linear(512, 128, BATCH_SIZE));
	net2.addLayer(new Tanh(128, BATCH_SIZE));

	net2.addLayer(new Linear(128, 10, BATCH_SIZE));
	net2.addLayer(new Tanh(10, BATCH_SIZE));

	net2.fit(plants.images, plants.labels, 0.001, 3);

	return -1;

	Mnist mnist(std::string("C:\\MNIST\\train-images.idx3-ubyte"), std::string("C:\\MNIST\\train-labels.idx1-ubyte"), BATCH_SIZE, 60000);
	mnist.print(1, 0);

	Network net;
	net.addLayer(new Linear(28 * 28, 128, BATCH_SIZE));
	net.addLayer(new Tanh(128, BATCH_SIZE));
	net.addLayer(new Linear(128, 10, BATCH_SIZE));
	net.addLayer(new Tanh(10, BATCH_SIZE));

	net.fit(mnist.images, mnist.labels, 0.01, 10);

	Mnist mnist_test(std::string("C:\\MNIST\\t10k-images.idx3-ubyte"), std::string("C:\\MNIST\\t10k-labels.idx1-ubyte"), BATCH_SIZE, 10000);
	net.evaluate(mnist_test.images, mnist_test.labels);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

