#include "hip/hip_runtime.h"
#include "MSE.h"
#include <stdexcept>
#include <iostream>

__global__ void cudaMSELoss(const float* predictions, const float* targets, float* loss, int size) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size) {
		float diff = predictions[idx] - targets[idx];
		atomicAdd(loss, diff * diff);
	}
}

__global__ void cudaMSELossDerivative(float* predictions, float* targets, float* derivatives, int size, int batch_size)
{
	//printf("cudaMSELossDerivative\r\n");

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < batch_size)
	{
		derivatives[idx] = 2.0 * (predictions[idx] - targets[idx]) / size;
	}
}

float MSE::cost(Tensor& y_pred, Tensor& y_real)
{
	if (y_pred.b_size() != y_real.b_size()) {
		throw std::invalid_argument("Wrong set sizes. Cannot perform fit.");
	}

	float* loss_dev;
	float loss = -13.0f;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc(&loss_dev, sizeof(float));
	cudaStatus = hipMemset(loss_dev, 0, sizeof(float));

	int size = y_real.rows * y_real.cols;
	int blockSize = size;
	int gridSize = (size + blockSize) / blockSize;

	cudaMSELoss << <gridSize, blockSize >> > (y_pred.dev, y_real.dev, loss_dev, size);
	cudaStatus = hipGetLastError();

	hipMemcpy(&loss, loss_dev, sizeof(float), hipMemcpyDeviceToHost);
	hipFree(loss_dev);

	return loss / y_real.cols / y_real.rows;
}

Tensor * MSE::derivative(Tensor& y_pred, Tensor& y_real)
{
	if (y_pred.b_size() != y_real.b_size()) {
		throw std::invalid_argument("Wrong set sizes. Cannot perform fit.");
	}

	int size = y_real.cols;
	int batch_size = y_real.rows;
	int blockSize = (size * batch_size);
	int gridSize = (size + blockSize) / blockSize;

	Tensor * deriv = new Tensor(y_real.rows, y_real.cols);

	cudaMSELossDerivative << <gridSize, blockSize >> > (y_pred.dev, y_real.dev, deriv->dev, size, batch_size*size);

	return deriv;
}
