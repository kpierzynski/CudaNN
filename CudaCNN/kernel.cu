﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "Tensor.h"
#include "Network.h"
#include "Linear.h"
#include "Tanh.h"
#include "MNISTSet.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

int main()
{
	Network net;
	net.addLayer(new Linear(28 * 28, 30));
	net.addLayer(new Tanh(30));
	net.addLayer(new Linear(30, 10));
	net.addLayer(new Tanh(10));

	MNISTSet mnist(std::string("C:\\MNIST\\train-images.idx3-ubyte"), std::string("C:\\MNIST\\train-labels.idx1-ubyte"), 60000);
	mnist.print(0);
	std::cout << "Fitting" << std::endl;
	net.fit(mnist.images, mnist.labels, 0.01f, 5);

	MNISTSet mnist_test(std::string("C:\\MNIST\\t10k-images.idx3-ubyte"), std::string("C:\\MNIST\\t10k-labels.idx1-ubyte"), 10000);
	net.evaluate(mnist_test.images, mnist_test.labels);

	return 0;
}
